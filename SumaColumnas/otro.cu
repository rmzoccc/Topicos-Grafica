
#include <hip/hip_runtime.h>
#include <iostream>
#include <ctime>
using namespace std;

__global__ void SumaColMatrizKernel(int f,int c,float*Md,float*Nd){
	float Pvalue=0;
	for(int k=threadIdx.x;k<f*c;k+=c){
		Pvalue=Pvalue+Md[k];
	}
	Nd[threadIdx.x]=Pvalue;
}

void SumaColMatrizHost(int f,int c,float*Mh){
	float *P;
	P=new float[c];
	for (int j=0;j<c;j++){
		P[j]=0;
		for (int i=0;i<f;i++){
			P[j]+=Mh[i*c+j];
		}
	}    
	cout<<"\nResultados HOST:"<<endl;
	for (int j=0;j<c;j++)	cout<<P[j]<<" ";
}


int main(){
	unsigned t0, t1, t2, t3;
	int f=10,c=2;
	cout<<"Filas: "<<f<<endl;
	cout<<"Columnas: "<<c<<endl;
	int size=f*c*sizeof(float);
	int size2=c*sizeof(float);

	//Guardando memoria en el host
	float *Mh=(float*)malloc(size);
	float *Nh=(float*)malloc(size2);

	cout<<"Matriz: ";
	for (int i=0;i<f*c;i++){
		Mh[i]=i+1;
		cout<<Mh[i]<<" ";
	}

	//Guarda memoria en el GPU
	float *Md,*Nd;
	hipMalloc(&Md,size);
	hipMalloc(&Nd,size2);

	hipMemcpy(Md, Mh, size, hipMemcpyHostToDevice);
	hipMemset(Nd, 0, size2);

	//Suma columnas en GPU
	int bloques=f/128+1;
	dim3 tamGrid(bloques,1);
	dim3 tamBlock(128,1,1);
	
	t0 = clock();
	SumaColMatrizKernel<<<tamGrid, tamBlock>>>(f,c,Md,Nd);
	t1 = clock();
	hipMemcpy(Nh, Nd, size2, hipMemcpyDeviceToHost);
	
	//Suma columnas en HOST
	t2 = clock();
	SumaColMatrizHost(f,c,Mh);
	t3 = clock();

	hipFree(Md);    hipFree(Nd);

	cout<<"\nResultados GPU: "<<endl;
	for(int i=0;i<c;i++){
		cout<<Nh[i]<<" ";
	}
	double time = (double(t1 - t0) / CLOCKS_PER_SEC);
   	cout << "Tiempo de ejecución en paralelo: " << time << endl;
	double time1 = (double(t3 - t2) / CLOCKS_PER_SEC);
   	cout << "Tiempo de ejecución en serie: " << time1 << endl;

}
