
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void SumaColMatrizKernel(int f,int c,float*Md,float*Nd){
	float Pvalue=0;
	for(int k=threadIdx.x;k<f*c;k+=c){
		Pvalue=Pvalue+Md[k];
	}
	Nd[threadIdx.x]=Pvalue;
}

void SumaColMatrizHost(int f,int c,float*Mh){
	float *P;
	P=new float[c];
	for (int j=0;j<c;j++){
		P[j]=0;
		for (int i=0;i<f;i++){
			P[j]+=Mh[i*c+j];
		}
	}    
	cout<<"\nResultados HOST:"<<endl;
	for (int j=0;j<c;j++)	cout<<P[j]<<" ";
}


int main(){
	int f=10,c=2;
	cout<<"Filas: "<<f<<endl;
	cout<<"Columnas: "<<c<<endl;
	int size=f*c*sizeof(float);
	int size2=c*sizeof(float);

	//Guardando memoria en el host
	float *Mh=(float*)malloc(size);
	float *Nh=(float*)malloc(size2);

	cout<<"Matriz: ";
	for (int i=0;i<f*c;i++){
		Mh[i]=i+1;
		cout<<Mh[i]<<" ";
	}

	//Guarda memoria en el GPU
	float *Md,*Nd;
	hipMalloc(&Md,size);
	hipMalloc(&Nd,size2);

	hipMemcpy(Md, Mh, size, hipMemcpyHostToDevice);
	hipMemset(Nd, 0, size2);

	//Suma columnas en GPU
	int bloques=f/128+1;
	dim3 tamGrid(bloques,1);
	dim3 tamBlock(128,1,1);

	SumaColMatrizKernel<<<tamGrid, tamBlock>>>(f,c,Md,Nd);
	hipMemcpy(Nh, Nd, size2, hipMemcpyDeviceToHost);

	//Suma columnas en HOST
	SumaColMatrizHost(f,c,Mh);


	hipFree(Md);    hipFree(Nd);

	cout<<"\nResultados GPU: "<<endl;
	for(int i=0;i<c;i++){
		cout<<Nh[i]<<" ";
	}

}
